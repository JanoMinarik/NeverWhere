#include "hip/hip_runtime.h"
#include <iostream>
#include "ccalc.h"
#include "grid.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define IDNX(i,j,ld) (i*ld + j)

void freeMem(double *cd_DM, double *cd_gVal, double *cd_wght, double *cd_gDns){
  hipFree( cd_DM );
  hipFree( cd_gVal );
  hipFree( cd_wght );
  hipFree( cd_gDns );
}

// calculation
__global__ void calcDens(int start, int pts, int noAOs, double *cd_DM, double *cd_gVal, double *cd_wght, double *cd_gDns){
  int tid = blockIdx.x;
  int k, l;

  if( tid + start < pts ){
    cd_gDns[ start + tid ] = 0.0;  
    for(k=0; k<noAOs; k++){
      for(l=0; l<k; l++){
        cd_gDns[ start + tid ]+= 2*cd_DM[ IDNX(k,l,noAOs) ]*cd_gVal[ (start+tid)*noAOs + k ]*cd_gVal[ (start+tid)*noAOs + l ];
      }
      cd_gDns[ start + tid ] += cd_DM[ IDNX(k,k,noAOs) ]*cd_gVal[ IDNX(start+tid, k, noAOs) ];
    }
    cd_gDns[ start + tid ] *= cd_wght[ start + tid ];
  }
}

void calcDensFast( int cores, grid myGrid ){
  //hipError_t cudaStat;
  //hipblasStatus_t cblStat;
  hipblasHandle_t handle;

  int pts, noAOs;
  double *cd_DM, *cd_gVal, *cd_wght, *cd_gDns;

  pts = myGrid.noPoints; noAOs = myGrid.noAOs;

  hipMalloc( (void**) &cd_DM, noAOs*noAOs*sizeof(double) );
  hipMalloc( (void**) &cd_gVal, pts*noAOs*sizeof(double) );
  hipMalloc( (void**) &cd_wght, pts*sizeof(double) );
  hipMalloc( (void**) &cd_gDns, pts*sizeof(double) );

  if(cd_DM == NULL || cd_gVal == NULL || cd_wght == NULL || cd_gDns == NULL){
    std::cout << " == Device memory allocation failed. ==\n";
    freeMem(cd_DM, cd_gVal, cd_wght, cd_gDns);
    return;
  }

  hipblasCreate(&handle);
  if(handle == NULL){
    std::cout << " == Could not initialize CUBLAS ==\n";
    freeMem(cd_DM, cd_gVal, cd_wght, cd_gDns);
    return;
  }

}

// interfaces
void calcDensCuda( int cores, grid myGrid ){
  hipError_t cs;
  double *cd_DM, *cd_gVal, *cd_wght, *cd_gDns; 
  
  hipMalloc( (void**) &cd_DM, myGrid.noAOs*myGrid.noAOs*sizeof(double) );
  hipMalloc( (void**) &cd_gVal, myGrid.noPoints*myGrid.noAOs*sizeof(double) );
  hipMalloc( (void**) &cd_gDns, myGrid.noPoints*sizeof(double) );
  hipMalloc( (void**) &cd_wght, myGrid.noPoints*sizeof(double) );

  if( cd_DM == NULL || cd_gVal == NULL || cd_gDns == NULL || cd_wght == NULL ){
    std::cout << " == ERROR CUDA memory allocation failed. ==\n";
    freeMem(cd_DM, cd_gVal, cd_wght, cd_gDns);
    return;
  }
 
  cs = hipMemcpy( cd_DM, myGrid.densityMatrix, myGrid.noAOs*myGrid.noAOs*sizeof(double), hipMemcpyHostToDevice );
  if(cs){std::cout << " Density Matrix data copy failed.\n";}
  double *pGVal = &myGrid.gridValue[0][0];
  cs = hipMemcpy( cd_gVal, (double*)&myGrid.gridValue[0][0], myGrid.noPoints*myGrid.noAOs*sizeof(double), hipMemcpyHostToDevice );
  if(cs){std::cout << " Grid Values data copy failed.\n";}
  cs = hipMemcpy( cd_wght, myGrid.weight, myGrid.noPoints*sizeof(double), hipMemcpyHostToDevice );
  if(cs){std::cout << " Weights data copy failed.\n";}

  for(int i=0; i<myGrid.noPoints; i+=cores){
    calcDens<<<cores,1>>>(i, cores, myGrid.noAOs, cd_DM, cd_gVal, cd_wght, cd_gDns);
  } 

  hipMemcpy( myGrid.gridDensity, cd_gDns, myGrid.noPoints*sizeof(double), hipMemcpyDeviceToHost );
  
  myGrid.atomDensity = 0.0;
  for(int i=0; i<myGrid.noPoints; i++){
    myGrid.atomDensity += myGrid.gridDensity[i]; 
  }

  freeMem(cd_DM, cd_gVal, cd_wght, cd_gDns);
}

